#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "header.h"


__global__  void checkIfMatch(int *img_cuda, int *obj_cuda, int numElements, int imgDim, int objDim, int startI, int startJ, double threshold, double *dev_res) {
    int thread = blockDim.x * blockIdx.x + threadIdx.x;


    // Increment the proper value of the arrray according to thread ID 
    if (thread < numElements){
        int i = thread / objDim;
        int j = thread % objDim;
        double p = img_cuda[(startI + i) * imgDim + (startJ + j)];
        double o = obj_cuda[i*objDim + j];
        dev_res[thread] = fabsf((p-o)/p);
    }
}

int isMatch(image_object img, image_object obj, int i, int j, double threshold){
    
    hipError_t err = hipSuccess;
    int dim2 = obj.dim * obj.dim;
    double *res = (double*)malloc(sizeof(double) * (img.dim * img.dim));
    // Allocate memory on GPU to copy the image from the host
    int *img_cuda;
    err = hipMalloc((void **)&img_cuda, img.dim * img.dim * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy image from host to the GPU memory
    err = hipMemcpy(img_cuda, img.matrix, img.dim * img.dim * sizeof(int), hipMemcpyHostToDevice);

    // Allocate memory on GPU to copy the object from the host
    int *obj_cuda;
    err = hipMalloc((void **)&obj_cuda, dim2 * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy object from host to the GPU memory
    err = hipMemcpy(obj_cuda, obj.matrix, dim2 * sizeof(int), hipMemcpyHostToDevice);

    double *dev_res;
    err = hipMalloc((void **)&dev_res, dim2 * sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =((dim2) + threadsPerBlock - 1) / threadsPerBlock;

    checkIfMatch<<<blocksPerGrid, threadsPerBlock>>>(img_cuda, obj_cuda, (dim2), img.dim, obj.dim, i, j, threshold, dev_res);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(res, dev_res, dim2 * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(img_cuda);
    hipFree(obj_cuda);
    hipFree(dev_res);

    int k, sum = 0;
    for (k = 0 ; k < dim2 ; k++){
        sum += res[k];
    }
    if(sum/(dim2) < threshold)
        return 1;
    return 0;
}